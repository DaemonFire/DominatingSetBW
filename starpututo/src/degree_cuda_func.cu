#include <starpu.h>

static void degree_compute_cuda(int n, int i, int *mat){
	unsigned j= blockIdx.x*blockDim.x + threadIdx.x;
	if ((j<n)&&(j>0))
		mat[i*n]+=mat[i*n+j];
}

extern "C" void degree_cuda_func (void *buffers[], void *_args){
	int n = STARPU_MATRIX_GET_NX(buffers[0]);
	int* i=(int*)_args;
	int *mat = (int*)STARPU_MATRIX_GET_PTR(buffers[0]);
	unsigned threads_per_block = 64;
	unsigned nblocks = (n + threads_per_block-1) / threads_per_block;

	degree_compute_cuda<<<nblocks, threads_per_block, 0, starpu_cuda_get_local_stream()>>>(n,*i,mat);
	hipStreamSynchronize(starpu_cuda_get_local_stream());
}
